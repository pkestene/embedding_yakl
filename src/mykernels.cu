#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void k_square(float *a,
                         int numElements)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    a[i] = a[i] * a[i];
  }
}


extern "C" void square(float *a,
                       int numElements)
{

  hipError_t err = hipSuccess;

  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  k_square<<<blocksPerGrid, threadsPerBlock>>>(a, numElements);
  err = hipGetLastError();
  hipDeviceSynchronize();

  if (err != hipSuccess)
    printf("Something bad happened !\n");
}